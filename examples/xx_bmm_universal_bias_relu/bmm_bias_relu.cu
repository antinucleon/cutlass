
#include <iostream>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"

#define CUTLASS_CHECK(status)                                             \
  {                                                                       \
    cutlass::Status error = status;                                       \
    if (error != cutlass::Status::kSuccess) {                             \
      std::cerr << "Got cutlass error: " << cutlassGetStatusString(error) \
                << " at: " << __LINE__ << std::endl;                      \
      exit(EXIT_FAILURE);                                                 \
    }                                                                     \
  }

// Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_nn_align2
using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_nn_align2 =
    cutlass::gemm::device::GemmUniversal<
        cutlass::half_t,
        cutlass::layout::ColumnMajor,
        cutlass::half_t,
        cutlass::layout::ColumnMajor,
        cutlass::half_t,
        cutlass::layout::RowMajor,
        float,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 64, 32>,
        cutlass::gemm::GemmShape<64, 32, 32>,
        cutlass::gemm::GemmShape<16, 8, 16>,

        cutlass::epilogue::thread::
            LinearCombination<cutlass::half_t, 2, float, float>,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
        6,
        2,
        2,

        cutlass::arch::OpMultiplyAdd

        >;

using BMMInstance =
    Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_nn_align2;

void bmm(cutlass::half_t* a_ptr,
         cutlass::half_t* b_ptr,
         cutlass::half_t* c_ptr,
         int* a_dim0,
         int* a_dim1,
         int* a_dim2,
         int* b_dim0,
         int* b_dim1,
         int* b_dim2,
         int* c_dim0,
         int* c_dim1,
         int* c_dim2) {
  int AB = *a_dim0;
  int AK = *a_dim1;
  int AM = *a_dim2;
  int BB = *b_dim0;
  int BN = *b_dim1;
  int BK = *b_dim2;
  int CB = AB;
  int CM = AM;
  int CN = BN;
  *c_dim0 = CB;
  *c_dim1 = CM;
  *c_dim2 = CN;
  const int B = AB;
  const int M = AM;
  const int N = BN;
  const int K = AK;

  if (B == 1024 && M == 128 && N == 30 && K == 752) {
    //  TODO: cast to right dtype
    using ElementComputeEpilogue = typename BMMInstance::ElementAccumulator;

    typename BMMInstance::Arguments arguments{

        cutlass::gemm::GemmUniversalMode::kBatched,
        {AM, BN, AK},
        AB,
        {ElementComputeEpilogue(1), ElementComputeEpilogue(0)},
        (void*)a_ptr,
        (void*)b_ptr,
        (void*)c_ptr,
        (void*)c_ptr,
        AK * AM,
        0,
        CM * CN,
        CM * CN,
        AM,
        BK,
        CN,
        CN

    };
    BMMInstance gemm_op;
    size_t workspace_size = gemm_op.get_workspace_size(arguments);
    // TODO: handle workspace correctly
    // This is a shitty code happened to be ok for non-split-k case
    // Because for fprop the workspace is 0
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
    auto status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);
    status = gemm_op.initialize(arguments, workspace.get());
    CUTLASS_CHECK(status);
    status = gemm_op();
    CUTLASS_CHECK(status);
    return;
  }
  throw std::runtime_error(
      "Unsupported workload for this conv2d specialization.");
}
int main(int argc, char** argv) {
  int B = std::atoi(argv[1]);
  int M = std::atoi(argv[2]);
  int N = std::atoi(argv[3]);
  int K = std::atoi(argv[4]);
  // This is a special bmm problem
  // X is [1, M, K]
  // W is [B. N, K]
  // Bias is [B, N]
  // Y is [B, M, N]

  int a_dim0 = B;
  int a_dim1 = K;
  int a_dim2 = M;
  int b_dim0 = 1;
  int b_dim1 = N;
  int b_dim2 = K;
  int c_dim0 = B;
  int c_dim1 = M;
  int c_dim2 = N;

  int AB = a_dim0;
  int AM = a_dim1;
  int AK = a_dim2;
  int BB = b_dim0;
  int BN = b_dim1;
  int BK = a_dim2;
  int CB = AB;
  int CM = AM;
  int CN = BN;

  using ElementOutput = typename GemmInstance::ElementC;
  using ElementInputA = typename GemmInstance::ElementA;
  using ElementInputB = typename GemmInstance::ElementB;

  cutlass::HostTensor<ElementInputA, typename GemmInstance::LayoutA> a(
      {a_dim0 * a_dim1, a_dim2});
  cutlass::HostTensor<ElementInputB, typename GemmInstance::LayoutB> b(
      {b_dim0 * b_dim1, b_dim2});
  cutlass::HostTensor<ElementOutput, typename GemmInstance::LayoutC> c(
      {c_dim0 * c_dim1, c_dim2});
  cutlass::HostTensor<ElementInputA, typename GemmInstance::LayoutA> bias(
      {c_dim2});

  // warmup

  bmm((cutlass::half_t*)a.device_data(), (cutlass::half_t*)b.device_data(),
      (cutlass::half_t*)bias.device_data(), (cutlass::half_t*)c.device_data(),
      &a_dim0, &a_dim1, &a_dim2, &b_dim0, &b_dim1, &b_dim2, &c_dim0, &c_dim1,
      &c_dim2);
  hipEvent_t events[2];
  for (auto& event : events) {
    hipEventCreate(&event);
  }
  hipEventRecord(events[0]);
  for (int i = 0; i < 5; ++i) {
    bmm((cutlass::half_t*)a.device_data(), (cutlass::half_t*)b.device_data(),
        (cutlass::half_t*)bias.device_data(), (cutlass::half_t*)c.device_data(),
        &a_dim0, &a_dim1, &a_dim2, &b_dim0, &b_dim1, &b_dim2, &c_dim0, &c_dim1,
        &c_dim2);
  }
  hipEventRecord(events[1]);
  hipEventSynchronize(events[1]);
  float runtime_ms = 0;
  hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }

  std::cout << runtime_ms << std::endl;
}